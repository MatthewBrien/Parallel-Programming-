#include "hip/hip_runtime.h"
// VectorAdd.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define N 100                                      // size of matricies
#define B 1                                      // blocks in the grid
#define T 100                                         // threads in a block

__global__ void multiply (int *a,int *b, int *c) {

      int index = threadIdx.y + blockDim *threadIdx.x;
      int row = threadIdx.x;
      int col = threadIdx.y;
        //int tidy = blockIdy.y *
        if(row < N  && col < N) {
          int i;
          double product = 0;
          for(i = 0; i< N; i++){
            product += a[row+i]*b[col*i];
          }
          c[index] = product;
        }
}
int main(void) {
    int a[N][N],b[N][N],c[N][N], d[N][N];
    int *dev_a, *dev_b, *dev_c;
    int matrix = N*N;  //user will enter a single dimension, but, it's like a square yo.
    double product;
    dim3 gridBlock(B,B,0); //2d structure of for the grid setup



    hipMalloc((void**)&dev_a,matrix * sizeof(int));
    hipMalloc((void**)&dev_b,matrix * sizeof(int));
    hipMalloc((void**)&dev_c,matrix * sizeof(int));

    //initialize the matrices
    int i, ii, iii;
    for (i=0;i<N;i++) {
      for(ii = 0; ii < N; ii++){
        a[i][ii] = i+ii;
        b[i][ii] = i+(2*ii);
      }
    }

    //copy values to gpu
    hipMemcpy(dev_a, a , N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b , N*sizeof(int),hipMemcpyHostToDevice);

    //call kernel
    multiply<<<B,gridBlock>>>(dev_a,dev_b,dev_c);

    hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

    //calculate sequentiallly
    for(i=0;i<N;i++){
      for(ii=0;ii<N;ii++){
        product = 0;
        for(iii=0;iii<N;iii++){
          product += a[i][iii] * b[iii][ii];

        }
        //compare sequential and parallel computations
        d[i][ii] = product;
        if(d[i][ii] != c[i][ii])
        {
          printf("Error at location %d, %d,   %f vs %f \n", i, ii, d[i][ii], c[i][ii]);
        }
      }
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
  return 0;
}
