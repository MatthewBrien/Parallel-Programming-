#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define N 17  // size of arrays
__global__ void transpose (int *a, int *b) {
                   int col = blockIdx.x*blockDim.x+threadIdx.x;
                   int row =blockIdx.y*blockDim.y+threadIdx.y;
                   int index1 = col + row * N;
                   int index2 = row + col * N;
                   if ( col < N && row < N) b[index1]= a[index2];
}
int main (int argc, char **argv ) {
 int i,j;
 int size = N * N *sizeof( int);
 int  a[N][N], *devA, *devB;
 int gridsize = (int) ceil((double) N/16);
for (i = 0; i < N; i++){                  // put some numbers into array
 for (j= 0; j < N; j++) {
    a[i][j] = j + N * i;
      }
    }

 printf("Initial values");
     for (i = 0; i < N; i++) {
      printf("\n");
      for (j= 0; j < N; j++) {
        printf("%3d  ",a[i][j]);
       }
      }

  printf("\nN = %d, grid size = %d\n",N,gridsize);
     dim3 block (16,16);
     dim3 grid (gridsize, gridsize);
     hipMalloc( (void**)&devA, size );
     hipMalloc( (void**)&devB, size );
     hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
     transpose<<<grid, block>>>(devA, devB);
     hipMemcpy( a, devB, size, hipMemcpyDeviceToHost);
         printf("Results");
        for (i = 0; i < N; i++) {
          printf("\n");
          for (j= 0; j < N; j++) {
            printf("%3d  ",a[i][j]);
          }
        }
 printf("\n");
 hipFree( devA);
 hipFree( devB);
 return (0);
}
